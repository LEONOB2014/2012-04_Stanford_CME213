#include "hip/hip_runtime.h"
/* This is machine problem 1, part 1, shift cypher
 *
 * The problem is to take in a string (a vector of characters) and a shift amount,
 * and add that number to each element of
 * the string, effectively "shifting" each element in the 
 * string.
 * 
 * We do this in four different ways:
 * 1. With a standard cuda kernel loading chars and outputting chars for each thread
 * 2. With a standard cuda kernel, casting the character pointer to an int so that
 *    we load and store 4 bytes each time instead of 1 which gives us better coalescing
 *    and uses the memory effectively to achieve higher bandwidth
 * 3. Same spiel except with a uint2, so that we load 8 bytes each time
 * 4. Similar to #1 except that we use thrust and don't write a kernel ourselves
 *
 */


#include <stdlib.h>
#include <stdio.h>
#include <ctime>
#include <fstream>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>

#include "mp1-util.h"


// Repeating from the tutorial, just in case you haven't looked at it.

// "kernels" or __global__ functions are the entry points to code that executes on the GPU
// The keyword __global__ indicates to the compiler that this function is a GPU entry point.
// __global__ functions must return void, and may only be called or "launched" from code that
// executes on the CPU.

void host_shift_cypher(std::vector<unsigned char> &input_array, std::vector<unsigned char> &output_array, unsigned char shift_amount)
{
  for(unsigned int i=0;i<input_array.size();i++)
  {
    unsigned char element = input_array[i];
    output_array[i] = element + shift_amount;
  }
}

// This kernel implements a per element shift
// by naively loading one byte and shifting it
__global__ void shift_cypher(unsigned char *input_array, unsigned char *output_array, unsigned char shift_amount,  unsigned int array_length)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < array_length)
        output_array[tid] = input_array[tid] + shift_amount;
}

//Here we load 4 bytes at a time instead of just 1
//to improve the bandwidth due to a better memory
//access pattern
__global__ void shift_cypher_int(unsigned int *input_array, unsigned int *output_array, unsigned int shift_amount, unsigned int array_length) 
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < array_length)
        output_array[tid] = input_array[tid] + shift_amount;
}

//Here we go even further and load 8 bytes
//does it make a further improvement?
__global__ void shift_cypher_int2(uint2 *input_array, uint2 *output_array, unsigned int shift_amount, unsigned int array_length) 
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < array_length) {
        output_array[tid].x = input_array[tid].x + shift_amount;
        output_array[tid].y = input_array[tid].y + shift_amount;
    }
}

bool checkResults(std::vector<unsigned char> &cipher_text_host, unsigned char *device_output_array,
                  const char *type) {
    //allocate space on host for gpu results
    std::vector<unsigned char> cipher_text_from_gpu(cipher_text_host.size());

    event_pair timer;
    start_timer(&timer);
    // download and inspect the result on the host:
    hipMemcpy(&cipher_text_from_gpu[0], device_output_array, cipher_text_host.size(), hipMemcpyDeviceToHost);
    check_launch("copy from gpu");
    stop_timer(&timer,"copy from gpu");
  
    // check CUDA output versus reference output
    int error = 0;
    for(int i=0;i<cipher_text_host.size();i++)
    {
        if(cipher_text_host[i] != cipher_text_from_gpu[i]) 
        { 
            error = 1;
            printf("Error at pos: %d\nexpected: %d got: %d\n", i, (int)cipher_text_host[i], (int)cipher_text_from_gpu[i]);
            break;
        }
    }

    if(error)
    {
        printf("Output of CUDA %s version and host version didn't match! \n", type);
        return false;
    }

    return true;
}

int main(void)
{

  //First load the text 
  std::ifstream ifs("mobydick.txt", std::ios::binary);
  if (!ifs.good()) {
      std::cerr << "Couldn't open book file!" << std::endl;
      return 1;
  }

  std::vector<unsigned char> text;

  ifs.seekg(0, std::ios::end); //seek to end of file
  int length = ifs.tellg();    //get distance from beginning
  ifs.seekg(0, std::ios::beg); //move back to beginning

  text.resize(length);
  ifs.read((char *)&text[0], length);

  ifs.close();

  //need to make a couple copies of the book, otherwise everything happens too quickly
  //make 2^4 = 16 copies
  for (int i = 0; i < 4; ++i) {
      text.insert(text.end(), text.begin(), text.end());
  }

  // compute the size of the arrays in bytes
  // with enough padding that a uint2 access won't be out of bounds
  int num_bytes = (text.size() + 7) * sizeof(unsigned char);

  //allocate host arrays
  std::vector<unsigned char> cipher_text_gpu(text.size());
  std::vector<unsigned char> cipher_text_host(text.size());

  // pointers to device arrays
  unsigned char *device_input_array  = 0;
  unsigned char *device_output_array = 0;
  
  event_pair timer;
  
  // hipMalloc device arrays
  hipMalloc((void**)&device_input_array,  num_bytes);
  hipMalloc((void**)&device_output_array, num_bytes);
  
  // if either memory allocation failed, report an error message
  if(device_input_array == 0 || device_output_array == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // generate random input string
  unsigned char shift_amount = (rand() % 25) + 1; //we don't want the shift to be 0!
  
  // do copies to and from gpu once to get rid of timing weirdness
  // on first time accesses due to driver
  // touch all memory
  hipMemcpy(device_input_array,  &text[0],            num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_output_array, device_input_array,  num_bytes, hipMemcpyDeviceToDevice);
  hipMemcpy(&text[0],            device_output_array, num_bytes, hipMemcpyDeviceToHost);

  start_timer(&timer);
  // copy input to GPU
  hipMemcpy(device_input_array, &text[0], num_bytes, hipMemcpyHostToDevice);
  check_launch("copy to gpu");
  stop_timer(&timer,"copy to gpu");
  
  // generate reference output
  {
      start_timer(&timer);
      host_shift_cypher(text, cipher_text_host, shift_amount);
      stop_timer(&timer,"host shift cypher");
  }

  // choose a number of threads per block
  // we use 512 threads here
  const int block_size = 512;

  bool noErrors = true;
  // generate GPU char output
  {
      int grid_size = (text.size() + block_size - 1) / block_size;
      start_timer(&timer);
      // launch kernel
      shift_cypher<<<grid_size,block_size>>>(device_input_array, device_output_array, shift_amount, text.size());
      check_launch("gpu shift cypher char");
      stop_timer(&timer,"gpu shift cypher char");
      if (!checkResults(cipher_text_host, device_output_array, "char")) {
          noErrors = false;
      }
  }

  // generate GPU uint output
  {
      int grid_size = ( (text.size() + 3)/4 + block_size - 1) / block_size;
      unsigned int iShift = (shift_amount | (shift_amount << 8) | (shift_amount << 16) | (shift_amount << 24));
      start_timer(&timer);
      // launch kernel
      shift_cypher_int<<<grid_size,block_size>>>((unsigned int *)device_input_array, (unsigned int *)device_output_array, iShift, (text.size() + 3)/4);
      check_launch("gpu shift cypher uint");
      stop_timer(&timer,"gpu shift cypher uint");
      if (!checkResults(cipher_text_host, device_output_array, "uint")) {
          noErrors = false;
      }
  }

  //generate GPU uint2 output
  {
      int grid_size = ( (text.size() + 7)/8 + block_size - 1) / block_size;
      unsigned int iShift = (shift_amount | (shift_amount << 8) | (shift_amount << 16) | (shift_amount << 24));
      start_timer(&timer);
      // launch kernel
      shift_cypher_int2<<<grid_size,block_size>>>((uint2 *)device_input_array, (uint2 *)device_output_array, iShift, (text.size() + 7)/8);
      check_launch("gpu shift cypher uint2");
      stop_timer(&timer,"gpu shift cypher uint2");
      if (!checkResults(cipher_text_host, device_output_array, "uint2")) {
          noErrors = false;
      }
  }

  //generate GPU output with thrust
  {
      thrust::device_vector<unsigned char> dText = text;
      thrust::device_vector<unsigned char> dOutput(dText.size());
      start_timer(&timer);
      thrust::transform(dText.begin(), dText.end(), thrust::make_constant_iterator(shift_amount), dOutput.begin(), thrust::plus<unsigned char>());
      //don't need to check the launch because thrust does that for us
      stop_timer(&timer,"gpu shift cypher thrust");
      if (!checkResults(cipher_text_host, thrust::raw_pointer_cast(&dOutput[0]), "thrust")) {
          noErrors = false;
      }
  }

  if (noErrors) {
      printf("All CUDA Versions matched reference output.  Outputting ciphered text.\n");
      std::ofstream ofs("mobydick_enciphered.txt");
      //use the original length, before we made copies
      for (int i = 0; i < length; ++i) {
          ofs << cipher_text_host[i];
      }
      ofs.close();
  }
 
  // deallocate memory
  hipFree(device_input_array);
  hipFree(device_output_array);
}

