#include "hip/hip_runtime.h"
/* This is machine problem 1, part 2
 *
 *            Page Ranking
 *
 * The problem is to compute the rank of a set of webpages
 * given a link graph, aka a graph where each node is a webpage,
 * and each edge is a link from one page to another.
 * We're going to use the Pagerank algorithm (http://en.wikipedia.org/wiki/Pagerank),
 * specifically the iterative algorithm for calculating the rank of a page
 * We're going to run 20 iterations of the propage step.
 * The CPU implementation is provided.  Write the CUDA version.
 * Fill in all the places marked by TODO.
 * Keep all function interfaces intact, don't change any existing
 * variable names.
 * 
 * Your results are automatically checked against the CPU, make sure
 * they match.
 *
 * From the time taken, calculate the achieved bandwidth (for the default
 * of avg_edges=8).  How does this compare to the bandwidth from part 1?
 * Explain any differences.
 * Does changing the block_size make any difference? Why?
 *
 * Make a plot of bandwidth vs. avg_edges [2, 20], explain the shape
 * of the curve.
 *
 * Does the size of the vector have a significant impact on the bandwidth?
 * 
 */

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <ctime>
#include <limits>
#include <iostream>
#include "mp1-util.h"

event_pair timer;

// amount of floating point numbers between answer and computed value 
// for the answer to be taken correctly. 2's complement magick.
const int maxUlps = 10;
  
void host_graph_propagate(unsigned int *graph_indices, unsigned int *graph_edges, float *graph_nodes_in, float *graph_nodes_out, float * inv_edges_per_node, int array_length)
{
   size_t nBytes=0;
  for(int i=0; i < array_length; i++)
  {
    float sum = 0.f; 
    nBytes += 2*sizeof(unsigned int);
    for(int j = graph_indices[i]; j < graph_indices[i+1]; j++)
    {
        nBytes += 2*sizeof(float);
        nBytes += 2*sizeof(unsigned int);

      sum += graph_nodes_in[graph_edges[j]]*inv_edges_per_node[graph_edges[j]];
    }
    nBytes += 1*sizeof(float);
    graph_nodes_out[i] = 0.5f/(float)array_length + 0.5f*sum;
  }
std::cout<<nBytes*20<<" ";
}


void host_graph_iterate(unsigned int *graph_indices, unsigned int *graph_edges, float *graph_nodes_A, float *graph_nodes_B, float * inv_edges_per_node, int nr_iterations, int array_length)
{
  //int nbytes = 293601072;
//size_t nBytes= 0;
  assert((nr_iterations % 2) == 0);
  for(int iter = 0; iter < nr_iterations; iter+=2)
  {
    host_graph_propagate(graph_indices, graph_edges, graph_nodes_A, graph_nodes_B, inv_edges_per_node, array_length);
    host_graph_propagate(graph_indices, graph_edges, graph_nodes_B, graph_nodes_A, inv_edges_per_node, array_length);
//nBytes = nBytes + 2*nbytes;
  }
//std::cout<<nBytes<<" ";
}

// TODO your kernel code here
__global__
void device_graph_propagate(unsigned int *graph_indices, unsigned int *graph_edges, float *graph_nodes_in, float *graph_nodes_out, float *inv_edges_per_node, int array_length)
{
  unsigned int i = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x ;
  if(i < array_length)
  {
    float sum = 0.f; 
    for(int j = graph_indices[i]; j < graph_indices[i+1]; j++)
    {
      sum += graph_nodes_in[graph_edges[j]]*inv_edges_per_node[graph_edges[j]];
    }
    graph_nodes_out[i] = 0.5f/(float)array_length + 0.5f*sum;
  }
}

//all of your gpu memory allocation and copying to gpu memory has to be in this function
void device_graph_iterate(unsigned int *h_graph_indices,
                          unsigned int *h_graph_edges,
                          float *h_graph_nodes_input,
                          float *h_graph_nodes_result,
                          float *h_inv_edges_per_node,
                          int nr_iterations,
                          int num_elements,
                          int avg_edges)
{
  unsigned int *d_graph_indices=0;
  unsigned int *d_graph_edges=0;
  float *d_graph_nodes_A=0;
  float *d_graph_nodes_B=0;
  float *d_inv_edges_per_node=0;

  //TODO allocate memory
  hipMalloc((void**)&d_graph_indices,  (num_elements+1) * sizeof(unsigned int));
  hipMalloc((void**)&d_graph_edges,  num_elements * avg_edges * sizeof(unsigned int));
  hipMalloc((void**)&d_graph_nodes_A, num_elements * sizeof(float));
  hipMalloc((void**)&d_graph_nodes_B, num_elements * sizeof(float));
  hipMalloc((void**)&d_inv_edges_per_node, num_elements * sizeof(float));
  if (d_graph_indices == 0 || d_graph_edges == 0 || d_graph_nodes_A == 0||
      d_graph_nodes_B == 0 || d_inv_edges_per_node == 0) {
      printf("Couldn't allocate enough memory on the GPU!\n");
      return;
  }

  //TODO copy memory to gpu
  hipMemcpy(d_graph_indices, &h_graph_indices[0], (num_elements+1) * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_graph_edges, &h_graph_edges[0], num_elements * avg_edges * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_inv_edges_per_node, &h_inv_edges_per_node[0], num_elements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_graph_nodes_A, &h_graph_nodes_input[0], num_elements * sizeof(float), hipMemcpyHostToDevice);
  start_timer(&timer);

int block_size = 128;

//for(int vl = 0; vl < 5; ++vl)
{
  //if(block_size<128)
     //block_size = block_size + 1;
  //else if(block_size < 256)
     //block_size = block_size + 32;
  //else if(block_size < 512)
     //block_size = block_size + 64;
//block_size *=2;
  // TODO your kernel calls
  int grid_size = (num_elements + block_size - 1)/block_size; 
      dim3 nthreads(block_size, 1, 1);
      // 2D grids
      dim3 nblocks(128, (grid_size + 127)/128);
//std::cout<<"\n"<<block_size<<" "<<grid_size<<" ";

  assert((nr_iterations % 2) == 0);
  for(int iter = 0; iter < nr_iterations; iter+=2)
  {
    device_graph_propagate<<<nblocks, nthreads>>>(d_graph_indices, d_graph_edges, d_graph_nodes_A, d_graph_nodes_B, d_inv_edges_per_node, num_elements);
    device_graph_propagate<<<nblocks, nthreads>>>(d_graph_indices, d_graph_edges, d_graph_nodes_B, d_graph_nodes_A, d_inv_edges_per_node, num_elements);
  }
  check_launch("gpu graph propagate");
  stop_timer(&timer,"gpu graph propagate");
}
  // TODO your final result should end up in h_graph_nodes_result, which is a *host* pointer
  hipMemcpy(&h_graph_nodes_result[0], d_graph_nodes_A, num_elements * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_graph_indices);
  hipFree(d_graph_edges);
  hipFree(d_graph_nodes_A);
  hipFree(d_graph_nodes_B);
  hipFree(d_inv_edges_per_node);
}


int main(void)
{
  // create arrays of 2M elements
  int num_elements = 1 << 21;
for(int vv=2;vv<21;vv++)
{
  int avg_edges = vv;
  int iterations = 20;
  
  // pointers to host & device arrays
  unsigned int *h_graph_indices = 0;
  float *h_inv_edges_per_node = 0;
  unsigned int *h_graph_edges = 0;
  float *h_graph_nodes_input = 0;
  float *h_graph_nodes_result = 0;
  float *h_graph_nodes_checker_A = 0;
  float *h_graph_nodes_checker_B = 0;
  
  // malloc host array
  // index array has to be n+1 so that the last thread can 
  // still look at its neighbor for a stopping point
  h_graph_indices         = (unsigned int*)malloc((num_elements+1) * sizeof(unsigned int));
  h_inv_edges_per_node    = (float*)       malloc((num_elements) * sizeof(float));
  h_graph_edges           = (unsigned int*)malloc(num_elements * avg_edges * sizeof(unsigned int));
  h_graph_nodes_input     = (float*)       malloc(num_elements * sizeof(float));
  h_graph_nodes_result    = (float*)       malloc(num_elements * sizeof(float));
  h_graph_nodes_checker_A = (float*)       malloc(num_elements * sizeof(float));
  h_graph_nodes_checker_B = (float*)       malloc(num_elements * sizeof(float));
  
  // if any memory allocation failed, report an error message
  if(h_graph_indices == 0 || h_graph_edges == 0 || h_graph_nodes_input == 0 || h_graph_nodes_result == 0 || 
	 h_inv_edges_per_node == 0 || h_graph_nodes_checker_A == 0 || h_graph_nodes_checker_B == 0)
  {
    printf("couldn't allocate memory\n");
    exit(1);
  }

  // generate random input
  // initialize
  srand(time(NULL));
   
  h_graph_indices[0] = 0;
  for(int i=0;i< num_elements;i++)
  {
    int nr_edges = (i % (2*avg_edges-1) + 1);
    h_inv_edges_per_node[i] = 1.f/(float)nr_edges;
    h_graph_indices[i+1] = h_graph_indices[i] + nr_edges;
    if(h_graph_indices[i+1] >= (num_elements * avg_edges))
    {
      printf("more edges than we have space for\n");
      exit(1);
    }
    for(int j=h_graph_indices[i];j<h_graph_indices[i+1];j++)
    {
      h_graph_edges[j] = rand() % num_elements;
    }
    
    h_graph_nodes_input[i] =  1.f/(float)num_elements;
    h_graph_nodes_checker_A[i] =  h_graph_nodes_input[i];
    h_graph_nodes_result[i] = std::numeric_limits<float>::infinity();
  }
  
  //do page rank on the GPU
  device_graph_iterate(h_graph_indices, h_graph_edges, h_graph_nodes_input, h_graph_nodes_result, h_inv_edges_per_node, iterations, num_elements, avg_edges);
  std::cout<<"\n";
  start_timer(&timer);
  // generate reference output on CPU
  host_graph_iterate(h_graph_indices, h_graph_edges, h_graph_nodes_checker_A, h_graph_nodes_checker_B, h_inv_edges_per_node, iterations, num_elements);
  
  check_launch("host graph propagate");
  stop_timer(&timer,"host graph propagate");
  
  // check CUDA output versus reference output
  int num_errors = 0;
  for(int i=0;i<num_elements;i++)
  {
    float n = h_graph_nodes_result[i];
    float c = h_graph_nodes_checker_A[i];
    if(!AlmostEqual2sComplement(n,c,maxUlps)) 
    {
      num_errors++;
      if (num_errors < 10)
      {
            printf("%d:%.16f::",i, n-c);
      }
    }
  }
  
  if(num_errors > 0)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else
  {
    printf("\n");
  }

  // deallocate memory
  free(h_graph_indices);
  free(h_inv_edges_per_node);
  free(h_graph_edges);
  free(h_graph_nodes_input);
  free(h_graph_nodes_result);
  free(h_graph_nodes_checker_A);
  free(h_graph_nodes_checker_B);
}

}

