#include "hip/hip_runtime.h"
/* Machine problem 2
 * 2D Heat Diffusion
 *
 * In this homework you will be implementing a finite difference 2D-Heat Diffusion Solver
 * in two different ways - with and without using shared memory.
 * You will impelement stencils of orders 2, 4 and 8.  A reference CPU implementation
 * has been provided.  You should keep all existing classes, method names, function names,
 * and variables as is.
 *
 * The simParams and Grid classes are provided for convenience. The simParams class will
 * load a file containing all the information needed for the simulation and calculate the
 * maximum stable CFL number.  The Grid will set up a grid with the appropriate boundary and
 * initial conditions.
 *
 * Some general notes about declaring N-dimensional arrays.
 * You may have seen / been taught to do this in the past:
 * int **A = (int **)malloc(numRows * sizeof(int *));
 * for (int r = 0; r < numRows; ++r)
 *     A[r] = (int *)malloc(numCols * sizeof(int));
 *
 * so that you can then access elements of A with the notation A[row][col], which involes dereferencing
 * two pointers.  This is a REALLY BAD way to represent 2D arrays for a couple of reasons.
 * 
 * 1) For a NxN array, it does N+1 mallocs which is slow.  And on the gpu setting up this data 
 *    structure is a pain in the butt.  But you _should_ know how to do it.
 * 2) There is absolutely no guarantee that different rows are even remotely close in memory;
 *    subsequent rows could allocated on complete opposite sides of the address space
 *    which leads to terrible cache characteristics
 * 3) The double indirection leads to really high memory latency.  To access location A[i][j],
 *    first we have to make a trip to memory to fetch A[i], and once we get that pointer, we have to make another
 *    trip to memory to fetch (A[i])[j].  It would be far better if we only had to make one trip to 
 *    memory.  This is _especially_ important on the gpu.
 *
 * The BETTER WAY - just allocate one 1-D array of size N*N.  Then just calculate the correct offset -
 * A[i][j] = *(A + i * numCols + j).  There is only one allocation, adjacent rows are as close as they can be
 * and we only make one trip to memory to fetch a value.  The grid implements this storage scheme 
 * "under the hood" and overloads the () operator to allow the more familiar (x, y) notation.
 *
 * For the GPU code in this exercise you don't need to worry about trying to be fancy and overload an operator
 * or use some #define macro magic to mimic the same behavior - you can just do the raw addressing calculations. 
 *
 * For the first part of the homework where you will implement the kernels without using shared memory
 * each thread should compute exactly one output and you should use 2D blocks that are 16x16.
 *
 * For the second part with shared memory - it is recommended that you use 1D blocks since the ideal
 * implementation will have each thread outputting more than 1 value and the addressing arithmetic
 * is actually easier with 1D blocks.
 *
 * Notice that the reference CPU computation and Grid class are templated so that they can be
 * declared with either float or double.  You are only required to implement a CUDA version 
 * with floats; impelementing a CUDA version with doubles is extra credit!
 */


#include <ostream>
#include <iostream>
#include <iomanip>
#include <limits>
#include <vector>
#include <fstream>
#include <string>
#include <assert.h>
#include <fstream>
#include <sstream>
#include <cmath>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "mp1-util.h"

class simParams {
    public:
        simParams(const char *filename, bool verbose); //parse command line
                                                       //does no error checking
        simParams(); //use some default values

        int    nx()         const {return nx_;}
        int    ny()         const {return ny_;}
        int    gx()         const {return gx_;}
        int    gy()         const {return gy_;}
        double lx()         const {return lx_;}
        double ly()         const {return ly_;}
        double alpha()      const {return alpha_;}
        int    iters()      const {return iters_;}
        double dx()         const {return dx_;}
        double dy()         const {return dy_;}
        double ic()         const {return ic_;}
        int    order()      const {return order_;}
        int    borderSize() const {return borderSize_;}
        double xcfl()       const {return xcfl_;}
        double ycfl()       const {return ycfl_;}
        double topBC()      const {return bc[0];}
        double leftBC()     const {return bc[1];}
        double bottomBC()   const {return bc[2];}
        double rightBC()    const {return bc[3];}

    private:
        int    nx_, ny_;     //number of grid points in each dimension
        int    gx_, gy_;     //number of grid points including halos
        double lx_, ly_;     //extent of physical domain in each dimension
        double alpha_;       //thermal conductivity
        double dt_;          //timestep
        int    iters_;       //number of iterations to do
        double dx_, dy_;     //size of grid cell in each dimension
        double ic_;          //uniform initial condition
        double xcfl_, ycfl_; //cfl numbers in each dimension
        int    order_;       //order of discretization
        int    borderSize_;  //number of halo points
        double bc[4];        //0 is top, counter-clockwise

        void calcDtCFL();
};

simParams::simParams() {
    nx_ = ny_ = 10;
    lx_ = ly_ = 1;
    alpha_ = 1;
    iters_ = 1000;
    order_ = 2;

    dx_ = lx_ / (nx_ - 1);
    dy_ = ly_ / (ny_ - 1);

    ic_ = 5.;

    bc[0] = 0.;
    bc[1] = 10.;
    bc[2] = 0.;
    bc[3] = 10.;

    calcDtCFL();

    borderSize_ = 0;
    if (order_ == 2)
        borderSize_ = 1;
    else if (order_ == 4)
        borderSize_ = 2;
    else if (order_ == 8)
        borderSize_ = 4;

    gx_ = nx_ + 2 * borderSize_;
    gy_ = ny_ + 2 * borderSize_;
}

simParams::simParams(const char *filename, bool verbose) {
    std::ifstream ifs(filename);

    if (!ifs.good()) {
        std::cerr << "Couldn't open parameter file!" << std::endl;
        exit(1);
    }

    ifs >> nx_ >> ny_;
    ifs >> lx_ >> ly_;
    ifs >> alpha_;
    ifs >> iters_;
    ifs >> order_;
    ifs >> ic_;
    ifs >> bc[0] >> bc[1] >> bc[2] >> bc[3];

    ifs.close();

    dx_ = lx_ / (nx_ - 1);
    dy_ = ly_ / (ny_ - 1);

    calcDtCFL();

    borderSize_ = 0;
    if (order_ == 2)
        borderSize_ = 1;
    else if (order_ == 4)
        borderSize_ = 2;
    else if (order_ == 8)
        borderSize_ = 4;

    gx_ = nx_ + 2 * borderSize_;
    gy_ = ny_ + 2 * borderSize_;

    if (verbose) {
        printf("nx: %d ny: %d\ngx: %d gy: %d\nlx %f: ly: %f\nalpha: %f\niterations: %d\norder: %d\nic: %f\n", 
                nx_, ny_, gx_, gy_, lx_, ly_, alpha_, iters_, order_, ic_);
        printf("dx: %f dy: %f\ndt: %f xcfl: %f ycfl: %f\n", 
                dx_, dy_, dt_, xcfl_, ycfl_);
    }
}

void simParams::calcDtCFL() {
    //check cfl number and make sure it is ok
    if (order_ == 2) {
        //make sure we come in just under the limit
        dt_ = (.5 - .0001) * (dx_ * dx_ * dy_ * dy_) / (alpha_ * (dx_ * dx_ + dy_ * dy_));
        xcfl_ = (alpha_ * dt_) / (dx_ * dx_);
        ycfl_ = (alpha_ * dt_) / (dy_ * dy_);
    }
    else if (order_ == 4) {
        dt_ = (.5 - .0001) * (12 * dx_ * dx_ * dy_ * dy_) / (16 * alpha_ * (dx_ * dx_ + dy_ * dy_));
        xcfl_ = (alpha_ * dt_) / (12 * dx_ * dx_);
        ycfl_ = (alpha_ * dt_) / (12 * dy_ * dy_);
    }
    else if (order_ == 8) {
        dt_ = (.5 - .0001) * (5040 * dx_ * dx_ * dy_ * dy_) / (8064 * alpha_ * (dx_ * dx_ + dy_ * dy_));
        xcfl_ = (alpha_ * dt_) / (5040 * dx_ * dx_);
        ycfl_ = (alpha_ * dt_) / (5040 * dy_ * dy_);
    }
    else {
        std::cerr << "Unsupported discretization order." << std::endl;
        exit(1);
    }
}

template<typename floatType>
class Grid {
    public:
        Grid(const simParams &params, bool debug);
        ~Grid() { }

        typedef int gridState;

        int gx() const {return gx_;}
        int gy() const {return gy_;}
        int nx() const {return nx_;}
        int ny() const {return ny_;}
        int borderSize() const {return borderSize_;}
        const gridState & curr() const {return curr_;}
        const gridState & prev() const {return prev_;}
        void swapState() {prev_ = curr_; curr_ ^= 1;} 

        //for speed doesn't do bounds checking
        floatType operator()(const gridState & selector, 
                                 int xpos, int ypos) const {
            return hGrid_[selector * gx_ * gy_ + ypos * gx_ + xpos];
        }

        floatType& operator()(const gridState & selector, 
                                  int xpos, int ypos) {
            return hGrid_[selector * gx_ * gy_ + ypos * gx_ + xpos];
        }

        void saveStateToFile(std::string identifier) const;
        std::vector<floatType> getGrid() const {return hGrid_;}

        template <class U> friend std::ostream & operator<<(std::ostream &os, const Grid<U>& grid);

    private:
        std::vector<floatType> hGrid_;

        int gx_, gy_;             //total grid extents
        int nx_, ny_;             //non-boundary region
        int borderSize_;          //number of halo cells

        gridState curr_;
        gridState prev_;

        bool debug_;

        //prevent copying and assignment since they are not implemented
        //and don't make sense for this class
        Grid(const Grid &);
        Grid& operator=(const Grid &);

};

template<typename floatType>
std::ostream& operator<<(std::ostream& os, const Grid<floatType> &grid) {
    os << std::setprecision(3);
    for (int y = grid.gy() - 1; y != -1; --y) {
        for (int x = 0; x < grid.gx(); x++) {
            os << std::setw(5) << grid(grid.curr(), x, y) << " ";
        }
        os << std::endl;
    }
    os << std::endl;
    return os;
}

template<typename floatType>
Grid<floatType>::Grid(const simParams &params, bool debug) {
    debug_ = debug;

    curr_ = 1;
    prev_ = 0;

    if (params.order() == 2) 
        borderSize_ = 1;
    else if (params.order() == 4)
        borderSize_ = 2;
    else if (params.order() == 8)
        borderSize_ = 4;

    ny_ = params.ny();
    nx_ = params.nx();

    assert(nx_ > 2 * borderSize_);
    assert(ny_ > 2 * borderSize_);

    gx_ = nx_ + 2 * borderSize_;
    gy_ = ny_ + 2 * borderSize_;
   
    if (debug) { 
        printf("(%d, %d) (%d, %d)\n", nx_, ny_, gx_, gy_);
    }

    //resize and set ICs
    hGrid_.resize(gx_ * gy_, params.ic());

    //set BCs
    for (int i = 0; i < gx_; ++i) {
        for (int j = 0; j < borderSize_; ++j) {
            (*this)(prev_, i, j) = params.bottomBC();
        }

        for (int j = 0; j < borderSize_; ++j) {
            (*this)(prev_, i, borderSize_ + ny_ + j) = params.topBC();
        }
    }

    for (int j = 0; j < gy_; ++j) {
        for (int i = 0; i < borderSize_; ++i) {
            (*this)(prev_, i, j) = params.leftBC();
        }

        for (int i = 0; i < borderSize_; ++i) {
            (*this)(prev_, borderSize_ + nx_ + i, j) = params.rightBC();
        }
    }

    //create the copy of the grid we need for ping-ponging
    hGrid_.insert(hGrid_.end(), hGrid_.begin(), hGrid_.end());
}

template<typename floatType>
void Grid<floatType>::saveStateToFile(std::string identifier) const {
    std::stringstream ss;
    ss << "grid" << "_" << identifier << ".txt";
    std::ofstream ofs(ss.str().c_str());
    
    ofs << *this << std::endl;

    ofs.close();
}

template <typename floatType>
inline floatType stencil2(const Grid<floatType> &grid, int x, int y, floatType xcfl, floatType ycfl, const typename Grid<floatType>::gridState &prev) {
    return grid(prev, x, y) + 
           xcfl * (grid(prev, x+1, y) + grid(prev, x-1, y) - 2 * grid(prev, x, y)) + 
           ycfl * (grid(prev, x, y+1) + grid(prev, x, y-1) - 2 * grid(prev, x, y));
}

template<typename floatType>
inline floatType stencil4(const Grid<floatType> &grid, int x, int y, floatType xcfl, floatType ycfl, const typename Grid<floatType>::gridState &prev) {
    return grid(prev, x, y) + 
           xcfl * (   -grid(prev, x+2, y) + 16 * grid(prev, x+1, y) -
                    30 * grid(prev, x, y) + 16 * grid(prev, x-1, y) - grid(prev, x-2, y)) + 
           ycfl * (   -grid(prev, x, y+2) + 16 * grid(prev, x, y+1) -
                    30 * grid(prev, x, y) + 16 * grid(prev, x, y-1) - grid(prev, x, y-2));
}

template<typename floatType>
inline floatType stencil8(const Grid<floatType> &grid, int x, int y, floatType xcfl, floatType ycfl, const typename Grid<floatType>::gridState &prev) {
    return grid(prev, x, y) + 
           xcfl * (   -9*grid(prev, x+4, y) + 128 * grid(prev, x+3, y) - 1008 * grid(prev, x+2, y) + 8064 * grid(prev, x+1, y) -
                                                     14350 * grid(prev, x, y) +
                      8064 * grid(prev, x-1, y) - 1008 * grid(prev, x-2, y) + 128 * grid(prev, x-3, y) -9 * grid(prev,x-4,y)) +
           ycfl * (   -9*grid(prev, x, y+4) + 128 * grid(prev, x, y+3) - 1008 * grid(prev, x, y+2) + 8064 * grid(prev,x, y+1) -
                                                     14350 * grid(prev, x, y) + 
                     8064 * grid(prev, x, y-1) - 1008 * grid(prev, x, y-2) + 128 * grid(prev, x, y-3) - 9 * grid(prev, x, y-4));
}

template <typename floatType>
void cpuComputation(Grid<floatType> &grid, const simParams &params) {
    std::string text;
    if (sizeof(floatType) == 4)
        text = "cpu computation float";
    else
        text = "cpu computation double";

    event_pair timer;
    start_timer(&timer);
    floatType xcfl = params.xcfl();
    floatType ycfl = params.ycfl();

    for (int i = 0; i < params.iters(); ++i) {
        grid.swapState();
        const typename Grid<floatType>::gridState& curr = grid.curr();
        const typename Grid<floatType>::gridState& prev = grid.prev();
        if (params.order() == 2) {
            for (int y = grid.borderSize(); y < grid.ny() + grid.borderSize(); ++y) {
                for (int x = grid.borderSize(); x < grid.nx() + grid.borderSize(); ++x) {
                    grid(curr, x, y) = stencil2(grid, x, y, xcfl, ycfl, prev);
                }
            }
        }
        else if (params.order() == 4) {
            for (int y = grid.borderSize(); y < grid.ny() + grid.borderSize(); ++y) {
                for (int x = grid.borderSize(); x < grid.nx() + grid.borderSize(); ++x) {
                    grid(curr, x, y) = stencil4(grid, x, y, xcfl, ycfl, prev);
                }
            }
        }
        else if (params.order() == 8) {
            for (int y = grid.borderSize(); y < grid.ny() + grid.borderSize(); ++y) {
                for (int x = grid.borderSize(); x < grid.nx() + grid.borderSize(); ++x) {
                    grid(curr, x, y) = stencil8(grid, x, y, xcfl, ycfl, prev);
                }
            }
        }
    }
    stop_timer(&timer, text.c_str());
}

__global__
void gpu2ndOrderStencil(float *curr, float *prev, int gx, int gy, int nx, int ny, float xcfl, float ycfl, int borderSize)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < borderSize || tidy < borderSize || tidx >= nx + borderSize || tidy >= ny + borderSize)
        return;

    int gl = tidy * gx + tidx;

    curr[gl] = prev[gl] + xcfl * (prev[gl+1]  + prev[gl-1]  - 2.f * prev[gl]) +
                          ycfl * (prev[gl-gx] + prev[gl+gx] - 2.f * prev[gl]);
}

__global__
void gpu4thOrderStencil(float *curr, float *prev, int gx, int gy, int nx, int ny, float xcfl, float ycfl, int borderSize)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < borderSize || tidy < borderSize || tidx >= nx + borderSize || tidy >= ny + borderSize)
        return;

    int gl = tidy * gx + tidx;

    curr[gl] = prev[gl] + xcfl * (-prev[gl+2] + 16.f * prev[gl+1]  - 30.f * prev[gl] 
                                              + 16.f * prev[gl-1] - prev[gl-2]) + 
                          ycfl * (-prev[gl+2*gx] + 16.f * prev[gl+gx] - 30.f * prev[gl] + 16.f * prev[gl-gx] - prev[gl-2*gx]);
}

__global__
void gpu8thOrderStencil(float *curr, float *prev, int gx, int gy, int nx, int ny, float xcfl, float ycfl, int borderSize)
{
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < borderSize || tidy < borderSize || tidx >= nx + borderSize || tidy >= ny + borderSize)
        return;

    int gl = tidy * gx + tidx;

    curr[gl] = prev[gl] + xcfl * (-9.f * prev[gl+4] + 128.f * prev[gl+3]  - 1008.f*prev[gl+2] + 8064.f*prev[gl+1] -
                                               14350.f * prev[gl] + 
                                  8064.f*prev[gl-1] - 1008.f*prev[gl-2] + 128.f*prev[gl-3] - 9.f*prev[gl-4]) +  
                          ycfl * (-9.f*prev[gl+4*gx] + 128.f*prev[gl+3*gx] - 1008.f*prev[gl+2*gx] + 8064.f*prev[gl+gx] - 
                                               14350.f * prev[gl] +
                                  8064.f*prev[gl-gx] - 1008.f*prev[gl-2*gx] + 128.f * prev[gl-3*gx] -9.f*prev[gl-4*gx]);
}

//because only the stencil computation differs between the different kernels
//we don't want to duplicate all that code, BUT we also don't want to introduce a ton of
//if statements inside the kernel.  So we use a template to compile only the correct code
template<int side, int usefulSide, int borderSize, int order>
__global__
void gpuShared(float *curr, float *prev, int gx, int gy, int nx, int ny, float xcfl, float ycfl)
{
    const int tid = threadIdx.x;

    const int xlane = tid % side;
    const int ylane = tid / side;

    const int numYPerStep = blockDim.x / side; //must divide evenly!

    __shared__ float smem[side * side];
    
    //how many blocks in x direction?
    int blocksXDir = (nx + usefulSide - 1) / usefulSide;

    int xBlock = blockIdx.x % blocksXDir;
    int yBlock = blockIdx.x / blocksXDir;

    //use threads to load our slice into smem
    for (int y = 0; y < (side + numYPerStep - 1) / numYPerStep; ++y) {
        int globalXPos = xBlock * usefulSide + xlane;
        int globalYPos = yBlock * usefulSide + y * numYPerStep + ylane;
        if (globalXPos < gx && globalYPos < gy)
            smem[(y * numYPerStep + ylane) * side + xlane] = prev[globalYPos * gx + globalXPos];
    }

    __syncthreads();

    //now that everything is loaded is smem, do the stencil calculation, we can store directly to global memory if we make sure to coalesce
    for (int y = borderSize; y < side - borderSize; y += numYPerStep) {
        //to make sure there aren't any bank conflicts, we need to leave a few of the processors idle
        if (xlane >= borderSize && xlane < side - borderSize && (y + ylane) < side - borderSize) {
            int xpos = xlane;
            int ypos = (y + ylane);

            int outputLocX = xBlock * usefulSide + xpos;
            int outputLocY = yBlock * usefulSide + ypos;
            if (outputLocX < gx - borderSize && outputLocY < gy - borderSize) {
                if (order == 2) { //compile time decision, not runtime
                    curr[outputLocY * gx + outputLocX] = smem[ypos * side + xpos] + 
                        xcfl * (smem[ypos * side + xpos + 1] + smem[ypos * side + xpos - 1] - 2.f * smem[ypos * side + xpos]) + 
                        ycfl * (smem[(ypos+1)*side + xpos] + smem[(ypos-1)*side + xpos] - 2.f * smem[ypos * side + xpos]);
                }
                else if (order == 4) {
                    curr[outputLocY * gx + outputLocX] = smem[ypos * side + xpos] + 
                        xcfl * (-smem[ypos * side + xpos + 2] + 16.f * smem[ypos * side + xpos + 1] - 
                                30.f * smem[ypos * side + xpos] + 16.f * smem[ypos * side + xpos - 1] - smem[ypos * side + xpos - 2]) + 
                        ycfl * (-smem[(ypos+2)*side + xpos] + 16.f * smem[(ypos+1)*side + xpos] - 
                                30.f*smem[ypos * side + xpos] + 16.f * smem[(ypos-1)*side + xpos] - smem[(ypos-2)*side+xpos]);
                }
                else if (order == 8) {
                    int basePos = ypos*side + xpos;
                    curr[outputLocY * gx + outputLocX] = smem[basePos] + 
                        xcfl*(-9.f*smem[basePos+4] + 128.f*smem[basePos+3] - 1008.f*smem[basePos+2] + 8064.f*smem[basePos+1] -
                                          14350.f*smem[basePos] +
                              8064.f*smem[basePos-1] - 1008.f*smem[basePos-2] + 128.f*smem[basePos-3] - 9.f*smem[basePos-4]) +
                       ycfl*(-9.f*smem[basePos+4*side] + 128.f*smem[basePos+3*side] - 1008.f*smem[basePos+2*side] + 8064.f*smem[basePos+side] -
                                          14350.f*smem[basePos] +
                              8064.f*smem[basePos-side] - 1008.f*smem[basePos-2*side] + 128.f*smem[basePos-3*side] - 9.f*smem[basePos-4*side]);
                }
            }
        }
    }
}

void gpuComputation(std::vector<float> &hInitialCondition, const simParams &params, std::vector<float> &hResults) {
    thrust::device_vector<float> dGridVec = hInitialCondition;
    float * dGrid = thrust::raw_pointer_cast(&dGridVec[0]);

    int totalSize = params.gx() * params.gy();
    dim3 threads(16, 16);
    dim3 blocks( (params.gx() + 15) / 16, (params.gy() + 15) / 16);
    int curr = 0;
    int prev = 1;
    event_pair timer;
    start_timer(&timer);
    if (params.order() == 2) {
        for (int i = 0; i < params.iters(); ++i) {
            prev = curr;
            curr ^= 1;
            gpu2ndOrderStencil<<<blocks, threads >>>(dGrid + curr * totalSize, dGrid + prev * totalSize, 
                    params.gx(), params.gy(), params.nx(), params.ny(), params.xcfl(), params.ycfl(), params.borderSize());
            check_launch("2ndOrderStencil");
        }
    }
    else if (params.order() == 4) {
        for (int i = 0; i < params.iters(); ++i) {
            prev = curr;
            curr ^= 1;
            gpu4thOrderStencil<<<blocks, threads >>>(dGrid + curr * totalSize, dGrid + prev * totalSize, 
                    params.gx(), params.gy(), params.nx(), params.ny(), params.xcfl(), params.ycfl(), params.borderSize());
            check_launch("4thOrderStencil");
        }
    }
    else if (params.order() == 8) {
        for (int i = 0; i < params.iters(); ++i) {
            prev = curr;
            curr ^= 1;
            gpu8thOrderStencil<<<blocks, threads >>>(dGrid + curr * totalSize, dGrid + prev * totalSize, 
                    params.gx(), params.gy(), params.nx(), params.ny(), params.xcfl(), params.ycfl(), params.borderSize());
            check_launch("8thOrderStencil");
        }
    }
    stop_timer(&timer, "gpu computation float");
    hResults.resize(totalSize);
    thrust::copy(dGridVec.begin() + curr * totalSize, dGridVec.end() - prev * totalSize, hResults.begin()); //only copy the last updated copy to the cpu
}

void gpuComputationShared8thOrder(std::vector<float> &hInitialCondition, const simParams &params, std::vector<float> &hResults) {
    thrust::device_vector<float> dGridVec = hInitialCondition;
    float * dGrid = thrust::raw_pointer_cast(&dGridVec[0]);

    const int borderSize = 4;
    assert(borderSize == params.borderSize()); //we hard code the borderSize so that we can use it with templates
                                               //but make sure that the value in the parameters agrees with us, just in case
    int totalSize = params.gx() * params.gy();
    int curr = 0;
    int prev = 1;

    event_pair timer;
    start_timer(&timer);

    for (int i = 0; i < params.iters(); ++i) {
        prev = curr;
        curr ^= 1;
        const int smemSide = 96;
        const int usefulsmemSide = smemSide - 2 * borderSize;
        int numBlocksX = (params.nx() + usefulsmemSide - 1) / usefulsmemSide;
        int numBlocksY = (params.ny() + usefulsmemSide - 1) / usefulsmemSide;
        gpuShared<smemSide, usefulsmemSide, borderSize, 8><<<numBlocksX * numBlocksY, 768 >>>(dGrid + curr * totalSize, dGrid + prev * totalSize, 
                                                                                              params.gx(), params.gy(), params.nx(), params.ny(), 
                                                                                              params.xcfl(), params.ycfl());
        check_launch("8thOrderStencil");
    }

    stop_timer(&timer, "shared gpu float");
    hResults.resize(totalSize);
    thrust::copy(dGridVec.begin() + curr * totalSize, dGridVec.end() - prev * totalSize, hResults.begin()); //only copy the last updated copy to the cpu
}

void gpuComputationShared4thOrder(std::vector<float> &hInitialCondition, const simParams &params, std::vector<float> &hResults) {
    thrust::device_vector<float> dGridVec = hInitialCondition;
    float * dGrid = thrust::raw_pointer_cast(&dGridVec[0]);

    const int borderSize = 2;
    assert(borderSize == params.borderSize()); //we hard code the borderSize so that we can use it with templates
                                               //but make sure that the value in the parameters agrees with us, just in case
    int totalSize = params.gx() * params.gy();
    int curr = 0;
    int prev = 1;

    event_pair timer;
    start_timer(&timer);

    for (int i = 0; i < params.iters(); ++i) {
        prev = curr;
        curr ^= 1;
        const int smemSide = 96;
        const int usefulsmemSide = smemSide - 2 * borderSize;
        int numBlocksX = (params.nx() + usefulsmemSide - 1) / usefulsmemSide;
        int numBlocksY = (params.ny() + usefulsmemSide - 1) / usefulsmemSide;
        gpuShared<smemSide, usefulsmemSide, borderSize, 4><<<numBlocksX * numBlocksY, 768 >>>(dGrid + curr * totalSize, dGrid + prev * totalSize, 
                                                                                              params.gx(), params.gy(), params.nx(), params.ny(), 
                                                                                              params.xcfl(), params.ycfl());
        check_launch("4thOrderStencil");
    }

    stop_timer(&timer, "shared gpu float");
    hResults.resize(totalSize);
    thrust::copy(dGridVec.begin() + curr * totalSize, dGridVec.end() - prev * totalSize, hResults.begin()); //only copy the last updated copy to the cpu
}

void gpuComputationShared2ndOrder(std::vector<float> &hInitialCondition, const simParams &params, std::vector<float> &hResults) {
    thrust::device_vector<float> dGridVec = hInitialCondition;
    float * dGrid = thrust::raw_pointer_cast(&dGridVec[0]);

    const int borderSize = 1;
    assert(borderSize == params.borderSize()); //we hard code the borderSize so that we can use it with templates
                                               //but make sure that the value in the parameters agrees with us, just in case
    int totalSize = params.gx() * params.gy();
    int curr = 0;
    int prev = 1;

    event_pair timer;
    start_timer(&timer);

    for (int i = 0; i < params.iters(); ++i) {
        prev = curr;
        curr ^= 1;
        const int smemSide = 96;
        const int usefulsmemSide = smemSide - 2 * borderSize;
        int numBlocksX = (params.nx() + usefulsmemSide - 1) / usefulsmemSide;
        int numBlocksY = (params.ny() + usefulsmemSide - 1) / usefulsmemSide;
        gpuShared<smemSide, usefulsmemSide, borderSize, 2><<<numBlocksX * numBlocksY, 768 >>>(dGrid + curr * totalSize, dGrid + prev * totalSize, 
                                                                                              params.gx(), params.gy(), params.nx(), params.ny(), 
                                                                                              params.xcfl(), params.ycfl());
        check_launch("2ndOrderStencil");
    }

    stop_timer(&timer, "shared gpu float");
    hResults.resize(totalSize);
    thrust::copy(dGridVec.begin() + curr * totalSize, dGridVec.end() - prev * totalSize, hResults.begin()); //only copy the last updated copy to the cpu
}

template<typename floatType>
void outputGrid(std::vector<floatType> &data, const simParams &params, std::string txt)
{
    std::stringstream ss;
    ss << "grid" << "_" << txt << ".txt";
    std::ofstream ofs(ss.str().c_str());
    
    ofs << std::setprecision(3);
    for (int y = params.gy() - 1; y != -1; --y) {
        for (int x = 0; x < params.gx(); x++) {
            ofs << std::setw(5) << data[y * params.gx() + x] << " ";
        }
        ofs << std::endl;
    }
    ofs << std::endl;

    ofs.close();
}

template <typename floatType>
int checkErrors(const Grid<floatType> &grid, const std::vector<floatType> &hGpuGrid, const simParams &params)
{
    //check that we got the same answer
    int error = 0;
    for (int x = params.borderSize(); x < params.gx() - params.borderSize(); ++x) {
        for (int y = params.borderSize(); y < params.gy() - params.borderSize(); ++y) {
            if (!AlmostEqual2sComplement(hGpuGrid[y * params.gx() + x], grid(grid.curr(), x, y), 10)) {
                if (error < 10) {
                    printf("Mis-match at pos: (%d, %d) cpu: %f, gpu: %f\n", x, y, grid(grid.curr(), x, y), hGpuGrid[y * params.gx() + x]);
                }
                ++error;
            }
        }
    }

    if (error)
        printf("There were %d total locations where there was a difference between the cpu and gpu\n", error);

    return error;
}


int main(int argc, char *argv[])
{
    if (argc != 2) {
        std::cerr << "Please supply a parameter file!" << std::endl;
        exit(1);
    }

    simParams params(argv[1], true);
    Grid<float> grid(params, true);

    grid.saveStateToFile("init"); //save our initial state, useful for making sure we
                                  //got setup and BCs right

    std::vector<float> hInitialCondition = grid.getGrid(); //make a copy of the initial state for the GPU
    std::vector<float> hInitialConditionShared = hInitialCondition;

    cpuComputation(grid, params);

    std::vector<float> hOutput;
    gpuComputation(hInitialCondition, params, hOutput);
    
    if (params.order() == 2)
        gpuComputationShared2ndOrder(hInitialConditionShared, params, hOutput);
    else if (params.order() == 4)
        gpuComputationShared4thOrder(hInitialConditionShared, params, hOutput);
    else if (params.order() == 8)
        gpuComputationShared8thOrder(hInitialConditionShared, params, hOutput);

    checkErrors(grid, hOutput, params);

    outputGrid(hOutput, params, "final_gpu");
    grid.saveStateToFile("final_cpu");

    return 0;
}
